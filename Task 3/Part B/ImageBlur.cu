#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"

/******************************************************************************

  Compile the program:
    nvcc -o ImageBlur ImageBlur.cu 

  Run the program:
	./ImageBlur

******************************************************************************/

__global__ void blur(unsigned char *gpu_image_output, unsigned char *gpu_image_input, int width, int height)
{

    int count = 1;

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    int i = blockIdx.x;
    int j = threadIdx.x;

    int t_red = 0;
    int t_green = 0;
    int t_blue = 0;
    int t_alpha = 0;

    if (i + 1 && j - 1)
    {

        int pos = blockDim.x * (blockIdx.x + 1) + threadIdx.x - 1;
        int pixel = pos * 4;

        t_red = gpu_image_input[idx * 4];
        t_green = gpu_image_input[idx * 4 + 1];
        t_blue = gpu_image_input[idx * 4 + 2];
        t_alpha = gpu_image_input[idx * 4 + 3];

        t_red += gpu_image_input[pixel];
        t_green += gpu_image_input[1 + pixel];
        t_blue += gpu_image_input[2 + pixel];
        t_alpha += gpu_image_input[3 + pixel];

        count++;
    }

    if (j + 1)
    {

        int pos = blockDim.x * (blockIdx.x) + threadIdx.x + 1;

        int pixel = pos * 4;

        t_red = gpu_image_input[idx * 4];
        t_green = gpu_image_input[idx * 4 + 1];
        t_blue = gpu_image_input[idx * 4 + 2];
        t_alpha = gpu_image_input[idx * 4 + 3];

        t_red += gpu_image_input[pixel];
        t_green += gpu_image_input[1 + pixel];
        t_blue += gpu_image_input[2 + pixel];
        t_alpha += gpu_image_input[3 + pixel];

        count++;
    }

    if (i + 1 && j + 1)
    {

        int pos = blockDim.x * (blockIdx.x + 1) + threadIdx.x + 1;

        int pixel = pos * 4;

        t_red = gpu_image_input[idx * 4];
        t_green = gpu_image_input[idx * 4 + 1];
        t_blue = gpu_image_input[idx * 4 + 2];
        t_alpha = gpu_image_input[idx * 4 + 3];

        t_red += gpu_image_input[pixel];
        t_green += gpu_image_input[1 + pixel];
        t_blue += gpu_image_input[2 + pixel];
        t_alpha += gpu_image_input[3 + pixel];

        count++;
    }

    if (i + 1)
    {

        int pos = blockDim.x * (blockIdx.x + 1) + threadIdx.x;

        int pixel = pos * 4;

        t_red = gpu_image_input[idx * 4];
        t_green = gpu_image_input[idx * 4 + 1];
        t_blue = gpu_image_input[idx * 4 + 2];
        t_alpha = gpu_image_input[idx * 4 + 3];

        t_red += gpu_image_input[pixel];
        t_green += gpu_image_input[1 + pixel];
        t_blue += gpu_image_input[2 + pixel];
        t_alpha += gpu_image_input[3 + pixel];

        count++;
    }

    if (j - 1)
    {

        int pos = blockDim.x * (blockIdx.x) + threadIdx.x - 1;

        int pixel = pos * 4;

        t_red = gpu_image_input[idx * 4];
        t_green = gpu_image_input[idx * 4 + 1];
        t_blue = gpu_image_input[idx * 4 + 2];
        t_alpha = gpu_image_input[idx * 4 + 3];

        t_red += gpu_image_input[pixel];
        t_green += gpu_image_input[1 + pixel];
        t_blue += gpu_image_input[2 + pixel];
        t_alpha += gpu_image_input[3 + pixel];

        count++;
    }

    if (i - 1)
    {

        int pos = blockDim.x * (blockIdx.x - 1) + threadIdx.x;

        int pixel = pos * 4;

        t_red = gpu_image_input[idx * 4];
        t_green = gpu_image_input[idx * 4 + 1];
        t_blue = gpu_image_input[idx * 4 + 2];
        t_alpha = gpu_image_input[idx * 4 + 3];

        t_red += gpu_image_input[pixel];
        t_green += gpu_image_input[1 + pixel];
        t_blue += gpu_image_input[2 + pixel];
        t_alpha += gpu_image_input[3 + pixel];

        count++;
    }

    int current_pixel = idx * 4;

    gpu_image_output[current_pixel] = t_red / count;
    gpu_image_output[1 + current_pixel] = t_green / count;
    gpu_image_output[2 + current_pixel] = t_blue / count;
    gpu_image_output[3 + current_pixel] = gpu_image_input[3 + current_pixel];
}

int main(int argc, char **argv)
{

    unsigned int error;
    unsigned int encError;
    unsigned char *image;
    unsigned int width;
    unsigned int height;
    const char *filename = "cat.png";
    const char *new_file_name = "blurred_cat.png";

    error = lodepng_decode32_file(&image, &width, &height, filename);
    if (error)
    {
        printf("error %u: %s\n", error, lodepng_error_text(error));
    }

    const int ARRAY_SIZE = width * height * 4;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

    unsigned char *host_image_input = (unsigned char *)malloc(ARRAY_SIZE * 4);
    unsigned char *host_image_output = (unsigned char *)malloc(ARRAY_SIZE * 4);

    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        host_image_input[i] = image[i];
    }

    // declare GPU memory pointers
    unsigned char *d_in;
    unsigned char *d_out;

    // allocate GPU memory
    hipMalloc((void **)&d_in, ARRAY_BYTES);
    hipMalloc((void **)&d_out, ARRAY_BYTES);

    hipMemcpy(d_in, host_image_input, ARRAY_BYTES, hipMemcpyHostToDevice);

    // launch the kernel
    blur<<<height, width>>>(d_out, d_in, width, height);

    // copy back the result array to the CPU
    hipMemcpy(host_image_output, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    encError = lodepng_encode32_file(new_file_name, host_image_output, width, height);
    if (encError)
    {
        printf("error %u: %s\n", error, lodepng_error_text(encError));
    }

    //free(image);
    //free(host_image_input);
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
