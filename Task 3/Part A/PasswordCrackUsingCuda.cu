
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

/******************************************************************************

  Compile the program:
    nvcc -o PasswordCrackUsingCuda PasswordCrackUsingCuda.cu 

  Run the program:
	./PasswordCrackUsingCuda > 3.1.txt

******************************************************************************/

__device__ int is_matched(char *password)
{
	char password_4[] = "CS2505";
	char password_2[] = "HP7719";
	char password_1[] = "MN0009";
	char password_3[] = "NY2021";

	char *p1 = password;
	char *p2 = password;
	char *p3 = password;
	char *p4 = password;

	char *w1 = password_1;
	char *w2 = password_2;
	char *w3 = password_3;
	char *w4 = password_4;

	while (*p1 == *w1)
	{

		if (*p1 == '\0')
		{

			printf("Password Found : %s\n", password_1);
			return 1;
		}
		p1++;
		w1++;
	}

	while (*p2 == *w2)
	{

		if (*p2 == '\0')
		{

			printf("Password Found : %s\n", password_2);
			return 1;
		}
		p2++;
		w2++;
	}

	while (*p3 == *w3)
	{

		if (*p3 == '\0')
		{

			printf("Password Found : %s\n", password_3);
			return 1;
		}
		p3++;
		w3++;
	}

	while (*p4 == *w4)
	{

		if (*p4 == '\0')
		{

			printf("Password Found : %s\n", password_4);
			return 1;
		}
		p4++;
		w4++;
	}
	return 0;
}

__global__ void kernel()
{

	char alphabets[26] = {'A', 'B', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'J', 'K', 'L', 'M', 'N', 'O', 'P', 'Q', 'S', 'T', 'U', 'V', 'W', 'X', 'Y', 'Z'};

	char numbers[10] = {'0', '1', '2', '3', '4', '5', '6', '7', '8', '9'};

	char password[7];
	password[6] = '\0';

	int a, b, c, d;

	for (a = 0; a < 10; a++)
	{

		for (b = 0; b < 10; b++)
		{

			for (c = 0; c < 10; c++)
			{

				for (d = 0; d < 10; d++)
				{

					password[0] = alphabets[blockIdx.x];
					password[1] = alphabets[threadIdx.x];
					password[2] = numbers[a];
					password[3] = numbers[b];
					password[4] = numbers[c];
					password[5] = numbers[d];

					if (is_matched(password))
					{

						printf("Password Found: %s\n", password);
					}
				}
			}
		}
	}
}

int time_difference(struct timespec *start, struct timespec *finish,
					long long int *difference)
{
	long long int ds = finish->tv_sec - start->tv_sec;
	long long int dn = finish->tv_nsec - start->tv_nsec;

	if (dn < 0)
	{
		ds--;
		dn += 1000000000;
	}
	*difference = ds * 1000000000 + dn;
	return !(*difference > 0);
}

int main(int argc, char *argv[])
{
	struct timespec start, finish;
	long long int time_elapsed;

	clock_gettime(CLOCK_MONOTONIC, &start);

	kernel<<<26, 26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time Elapsed: %lldns or %0.9lfs\n",
		   time_elapsed, (time_elapsed / 1.0e9));

	return 0;
}
